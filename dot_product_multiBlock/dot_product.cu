
#include <hip/hip_runtime.h>
#include <stdio.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

// Size of arrays
#define N 4096
#define THREADS_PER_BLOCK 511

// Kernel
__global__ void dot_prod(int *a, int *b, int *res, int n)
{
	__shared__ int products[THREADS_PER_BLOCK];

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	products[threadIdx.x] = a[id]*b[id];

	__syncthreads();

	if(threadIdx.x == 0)
	{
		int sum_of_products = 0;

		for(int i=0; i<THREADS_PER_BLOCK; i++)
		{
			sum_of_products = sum_of_products + products[i];
		}

		atomicAdd(res, sum_of_products);
	}
}

// Main program
int main()
{
	// Number of bytes to allocate for N integers
	size_t bytes = N*sizeof(int);

	// Allocate memory for arrays A, B, and result on host
	int *A = (int*)malloc(bytes);
	int *B = (int*)malloc(bytes);
	int *result = (int*)malloc(sizeof(int));

	// Allocate memory for arrays d_A, d_B, and d_result on device
	int *d_A, *d_B, *d_result;
	cudaErrorCheck( hipMalloc(&d_A, bytes) );
	cudaErrorCheck( hipMalloc(&d_B, bytes) );
	cudaErrorCheck( hipMalloc(&d_result, sizeof(int)) );

	// Fill host arrays A and B
  for(int i=0; i<N; i++)
  {
    A[i] = 1;
    B[i] = 2;
  }

  // Copy data from host arrays A and B to device arrays d_A and d_B
  cudaErrorCheck( hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice) );
  cudaErrorCheck( hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice) );

  // Set execution configuration parameters
  //    thr_per_blk: number of CUDA threads per grid block
  //    blk_in_grid: number of blocks in grid
  int thr_per_blk = THREADS_PER_BLOCK;
  int blk_in_grid = ceil( float(N) / thr_per_blk );

	printf("\nLaunching Grid:\n");
	printf("-----------------\n");
	printf("Threads Per Block: %d\n", thr_per_blk);
	printf("Blocks In Grid   : %d\n", blk_in_grid);
	printf("-----------------\n\n");

	// Launch kernel
	dot_prod<<< blk_in_grid, thr_per_blk >>>(d_A, d_B, d_result, N);

	  // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
  hipError_t cuErrSync  = hipGetLastError();

  // Check for errors on the GPU after control is returned to CPU
  hipError_t cuErrAsync = hipDeviceSynchronize();

  if (cuErrSync != hipSuccess) { printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrSync)); exit(0); }
  if (cuErrAsync != hipSuccess) { printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrAsync)); exit(0); }

	// Copy result from device to host
	cudaErrorCheck( hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost) );

	// Verify results
	if(*result != 2*N) { printf("Error: result is %d instead of %d\n", *result, 2*N); exit(0); }

	// Free CPU memory
	free(A);
	free(B);

	// Free GPU memory
	cudaErrorCheck( hipFree(d_A) );
	cudaErrorCheck( hipFree(d_B) );
	cudaErrorCheck( hipFree(d_result) );

	printf("__SUCCESS__\n");
	
	return 0;
}
