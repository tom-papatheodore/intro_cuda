
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024

__global__ void add_vectors(int *a, int *b, int *c, int n)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < n) c[id] = a[id] + b[id];
}

int main()
{
	// Number of bytes to allocate for N integers
	size_t bytes = N*sizeof(int);

  // Allocate memory for arrays A, B, and C on host
  int *A = (int*)malloc(bytes);
  int *B = (int*)malloc(bytes);
  int *C = (int*)malloc(bytes);

	// Allocate memory for arrays d_A, d_B, and d_C on device
	int *d_A, *d_B, *d_C;

	hipMalloc(&d_A, bytes);	
	hipMalloc(&d_B, bytes);
	hipMalloc(&d_C, bytes);

  // Fill host arrays A and B
  for(int i=0; i<N; i++)
  {
    A[i] = 1;
    B[i] = 2;
  }

	// Copy data from host arrays A and B to device arrays d_A and d_B
	hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

	// Set execution configuration parameters and launch kernel
	int threads_in_block = 128;
	int blocks_in_grid = ceil(float(N)/threads_in_block);
	add_vectors<<< blocks_in_grid, threads_in_block >>>(d_A, d_B, d_C, N);

	// Copy data from device array d_C to host array C
	hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

	// Verify results
	for(int i=0; i<N; i++)
	{
		if(C[i] != 3)
		{ 
			printf("Error: value of C[%d] = %d instead of 3\n", i, C[i]);
			exit(-1);
		}
	}	

	printf("__SUCCESS__\n");

	free(A);
	free(B);
	free(C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}
