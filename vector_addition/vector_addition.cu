
#include <hip/hip_runtime.h>
#include <stdio.h>

// Size of array
#define N 1048576

// Kernel
__global__ void add_vectors(int *a, int *b, int *c)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < N) c[id] = a[id] + b[id];
}

// Main program
int main()
{
	// Number of bytes to allocate for N integers
	size_t bytes = N*sizeof(int);

	// Allocate memory for arrays A, B, and C on host
	int *A = (int*)malloc(bytes);
	int *B = (int*)malloc(bytes);
	int *C = (int*)malloc(bytes);

	// Allocate memory for arrays d_A, d_B, and d_C on device
	int *d_A, *d_B, *d_C;
	hipMalloc(&d_A, bytes);
	hipMalloc(&d_B, bytes);
	hipMalloc(&d_C, bytes);

	// Fill host arrays A and B
	for(int i=0; i<N; i++)
	{
		A[i] = 1;
		B[i] = 2;
	}

	// Copy data from host arrays A and B to device arrays d_A and d_B
	hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

	// Set execution configuration parameters
	//		thr_per_blk: number of CUDA threads per grid block
	//		blk_in_grid: number of blocks in grid
	int thr_per_blk = 256;
	int blk_in_grid = ceil( float(N) / thr_per_blk );

	// Launch kernel
	add_vectors<<< blk_in_grid, thr_per_blk >>>(d_A, d_B, d_C);

	// Copy data from device array d_C to host array C
	hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

	// Verify results
	for(int i=0; i<N; i++)
	{
		if(C[i] != 3)
		{ 
			printf("\nError: value of C[%d] = %d instead of 3\n\n", i, C[i]);
			exit(-1);
		}
	}	

	// Free CPU memory
	free(A);
	free(B);
	free(C);

	// Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

  printf("\n---------------------------\n");
	printf("__SUCCESS__\n");
	printf("---------------------------\n");
	printf("N                 = %d\n", N);
	printf("Threads Per Block = %d\n", thr_per_blk);
	printf("Blocks In Grid    = %d\n", blk_in_grid);
  printf("---------------------------\n\n");

	return 0;
}
