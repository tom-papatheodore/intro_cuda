
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024

__global__ void add_vectors(int *a, int *b, int *c, int n)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < n) c[id] = a[id] + b[id];
}

int main()
{
	// Number of bytes to allocate for N integers
	size_t bytes = N*sizeof(int);

  // Allocate memory for arrays A, B, and C on host
  int *A = (int*)malloc(bytes);
  int *B = (int*)malloc(bytes);
  int *C = (int*)malloc(bytes);

	// Allocate memory for arrays d_A, d_B, and d_C on device
	int *d_A, *d_B, *d_C;

	hipMalloc(&d_A, bytes);	
	hipMalloc(&d_B, bytes);
	hipMalloc(&d_C, bytes);

  // Fill host arrays A and B
  for(int i=0; i<N; i++)
  {
    A[i] = 1;
    B[i] = 2;
  }

	// Copy data from host arrays A and B to device arrays d_A and d_B
	hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

	// Set execution configuration parameters and launch kernel
	//		thr_per_blk: number of CUDA threads per grid block
	//		blk_in_grid: number of blocks in grid
	int thr_per_blk = 128;
	int blk_in_grid = ceil( float(N) / thr_per_blk );

  add_vectors<<< blk_in_grid, thr_per_blk >>>(d_A, d_B, d_C, N);

	// Copy data from device array d_C to host array C
	hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

	// Verify results
	for(int i=0; i<N; i++)
	{
		if(C[i] != 3)
		{ 
			printf("Error: value of C[%d] = %d instead of 3\n", i, C[i]);
			exit(-1);
		}
	}	

	// Free CPU memory
	free(A);
	free(B);
	free(C);

	// Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	printf("__SUCCESS__\n");

	return 0;
}
